#include "hip/hip_runtime.h"
#include "utils.hpp"
#include "decodeLabel.hpp"
#include <hip/hip_runtime.h>
#include <stdexcept>
#include <iostream>
#include <stdio.h>
#include <limits>

// clang-format off
    #pragma GCC diagnostic push
    #pragma GCC diagnostic ignored "-Wdeprecated-declarations"
// clang-format on

__global__ void decodeLabelKernel(float* mask, float* maskLane, float* maskFreeSpace, float* image, const float* label,
                                const uint32_t width, const uint32_t height, float maxVal, const float* mean,
                                const float* stdDev)
{
    
    const uint8_t colors[36][3] = {
        {128,  64, 128},
        {250, 170, 160},
        { 81,   0,  81},
        {244,  35, 232},
        {230, 150, 140},
        {152, 251, 152},
        {220,  20,  60},
        {246, 198, 145},
        {255,   0,   0},
        {  0,   0, 230},
        {119,  11,  32},
        {255, 204,  54},
        {  0,   0, 142},
        {  0,   0,  70},
        {  0,  60, 100},
        {  0,   0,  90},
        {  0,   0, 110},
        {  0,  80, 100},
        {136, 143, 153},
        {220, 190,  40},
        {102, 102, 156},
        {190, 153, 153},
        {180, 165, 180},
        {174,  64,  67},
        {220, 220,   0},
        {250, 170,  30},
        {153, 153, 153},
        {153, 153, 153},
        {169, 187, 214},
        { 70,  70,  70},
        {150, 100, 100},
        {150, 120,  90},
        {107, 142,  35},
        { 70, 130, 180},
        {169, 187, 214},
        {0, 0, 0},
    };

    
    const uint32_t tidx = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t tidy = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidx >= width || tidy >= height)
        return;

    int maxIdx = -1;
    uint32_t startPos = (tidy * width) + tidx;
    for(int i=0; i<35; ++i)
    {
        // // (35, 480, 640)
        if (label[startPos + (i * height * width)] > maxVal)
        {
            maxIdx = i;
            maxVal = label[startPos + (i * height * width)];
            // printf(" %f ", maxVal);
        }
    }

    mask[tidy * width * 3 + 3 * tidx + 0] = colors[maxIdx][0];
    mask[tidy * width * 3 + 3 * tidx + 1] = colors[maxIdx][1];
    mask[tidy * width * 3 + 3 * tidx + 2] = colors[maxIdx][2];

    // preparing blended image for lane estimation
    float blend_ratio = 0.45;
    maskLane[tidy * width * 3 + 3 * tidx + 0] = (stdDev[0]*image[startPos + 0 * height * width] + mean[0]) * blend_ratio + mask[tidy * width * 3 + 3 * tidx + 0] * (1 - blend_ratio);
    maskLane[tidy * width * 3 + 3 * tidx + 1] = (stdDev[1]*image[startPos + 1 * height * width] + mean[1]) * blend_ratio + mask[tidy * width * 3 + 3 * tidx + 1] * (1 - blend_ratio);
    maskLane[tidy * width * 3 + 3 * tidx + 2] = (stdDev[2]*image[startPos + 2 * height * width] + mean[2]) * blend_ratio + mask[tidy * width * 3 + 3 * tidx + 2] * (1 - blend_ratio);

    // maskLane[tidy * width * 3 + 3 * tidx + 0] = image[startPos + 0 * height * width] * blend_ratio + mask[tidy * width * 3 + 3 * tidx + 0] * (1 - blend_ratio);
    // maskLane[tidy * width * 3 + 3 * tidx + 1] = image[startPos + 1 * height * width] * blend_ratio + mask[tidy * width * 3 + 3 * tidx + 1] * (1 - blend_ratio);
    // maskLane[tidy * width * 3 + 3 * tidx + 2] = image[startPos + 2 * height * width] * blend_ratio + mask[tidy * width * 3 + 3 * tidx + 2] * (1 - blend_ratio);

    // preparing color mask for free space
    if (maxIdx != 0)
        maxIdx = 35;

    maskFreeSpace[tidy * width * 3 + 3 * tidx + 0] = colors[maxIdx][0];
    maskFreeSpace[tidy * width * 3 + 3 * tidx + 1] = colors[maxIdx][1];
    maskFreeSpace[tidy * width * 3 + 3 * tidx + 2] = colors[maxIdx][2];
    
}

void batchedDecodeLabel(float* mask,  float* maskLane, float* maskFreeSpace, float* image, 
                        const float* gpuBuffer, const int batchSize, 
                        const int width, const int height, const float* mean,
                        const float* stdDev)
{
    for(int i=0; i < batchSize; ++i)
    {
        float maxVal = std::numeric_limits<float>::min();
        auto maskCUDA_ptr = mask + i*width*height*3;
        auto maskCUDA_Lane_ptr = maskLane + i*width*height*3;
        auto maskCUDA_FreeSpace_ptr = maskFreeSpace + i*width*height*3;
        auto image_ptr = image + i*width*height*3;
        auto gpuBuffer_ptr = gpuBuffer + i*width*height*35;

        dim3 numThreads = dim3(32, 32);

        // std::cout << "IN" << std::endl;

        decodeLabelKernel<<<dim3(iDivUp(width, numThreads.x),
                                 iDivUp(height, numThreads.y)),
                            numThreads>>>(maskCUDA_ptr,
                                        maskCUDA_Lane_ptr,
                                        maskCUDA_FreeSpace_ptr,
                                        image_ptr,
                                        gpuBuffer_ptr,
                                        width,
                                        height, 
                                        maxVal,
                                        mean,
                                        stdDev);
    }
}

// clang-format on
#pragma GCC diagnostic pop
// clang-format off